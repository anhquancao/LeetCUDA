#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
// #define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
// #define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
// #define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
// #define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
// #define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// -------------------------------------- FP32
// -------------------------------------- ElementWise Add grid(N/256),
// block(256) a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32_kernel(float *a, float *b, float *c,
                                           int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

// ElementWise Add + Vec4
// grid(N/256), block(256/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32x4_kernel(float *a, float *b, float *c,
                                             int N) {

}

// -------------------------------------- FP16
// -------------------------------------- ElementWise Add grid(N/256),
// block(256) a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f16_kernel(half *a, half *b, half *c, int N) {

}

// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f16x2_kernel(half *a, half *b, half *c, int N) {


}

__global__ void elementwise_add_f16x8_kernel(half *a, half *b, half *c, int N) {

}

__global__ void elementwise_add_f16x8_pack_kernel(half *a, half *b, half *c,
                                                  int N) {
}

// --------------------- PyTorch bindings for custom kernel
// -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define TORCH_BINDING_ELEM_ADD(packed_type, th_type, element_type, n_elements) \
  void elementwise_add_##packed_type(torch::Tensor a, torch::Tensor b,         \
                                     torch::Tensor c) {                        \
    CHECK_TORCH_TENSOR_DTYPE(a, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(b, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(c, (th_type))                                     \
    const int ndim = a.dim();                                                  \
    if (ndim != 2) {                                                           \
      int N = 1;                                                               \
      for (int i = 0; i < ndim; ++i) {                                         \
        N *= a.size(i);                                                        \
      }                                                                        \
      dim3 block(256 / (n_elements));                                          \
      dim3 grid((N + 256 - 1) / 256);                                          \
      elementwise_add_##packed_type##_kernel<<<grid, block>>>(                 \
          reinterpret_cast<element_type *>(a.data_ptr()),                      \
          reinterpret_cast<element_type *>(b.data_ptr()),                      \
          reinterpret_cast<element_type *>(c.data_ptr()), N);                  \
    } else {                                                                   \
      const int S = a.size(0);                                                 \
      const int K = a.size(1);                                                 \
      const int N = S * K;                                                     \
      if ((K / (n_elements)) <= 1024) {                                        \
        dim3 block(K / (n_elements));                                          \
        dim3 grid(S);                                                          \
        elementwise_add_##packed_type##_kernel<<<grid, block>>>(               \
            reinterpret_cast<element_type *>(a.data_ptr()),                    \
            reinterpret_cast<element_type *>(b.data_ptr()),                    \
            reinterpret_cast<element_type *>(c.data_ptr()), N);                \
      } else {                                                                 \
        int N = 1;                                                             \
        for (int i = 0; i < ndim; ++i) {                                       \
          N *= a.size(i);                                                      \
        }                                                                      \
        dim3 block(256 / (n_elements));                                        \
        dim3 grid((N + 256 - 1) / 256);                                        \
        elementwise_add_##packed_type##_kernel<<<grid, block>>>(               \
            reinterpret_cast<element_type *>(a.data_ptr()),                    \
            reinterpret_cast<element_type *>(b.data_ptr()),                    \
            reinterpret_cast<element_type *>(c.data_ptr()), N);                \
      }                                                                        \
    }                                                                          \
  }

TORCH_BINDING_ELEM_ADD(f32, torch::kFloat32, float, 1)
// TORCH_BINDING_ELEM_ADD(f32x4, torch::kFloat32, float, 4)
// TORCH_BINDING_ELEM_ADD(f16, torch::kHalf, half, 1)
// TORCH_BINDING_ELEM_ADD(f16x2, torch::kHalf, half, 2)
// TORCH_BINDING_ELEM_ADD(f16x8, torch::kHalf, half, 8)
// TORCH_BINDING_ELEM_ADD(f16x8_pack, torch::kHalf, half, 8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f32)
  // TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f32x4)
  // TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16)
  // TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16x2)
  // TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16x8)
  // TORCH_BINDING_COMMON_EXTENSION(elementwise_add_f16x8_pack)
}
